#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

/*****************************************************************************
 * This program checks if it is being run on a computer with a CUDA compatible
 * GPU, i.e. a modern nVidia GPU. If this program reports there are no
 * devices then no other programs in this set are going to work on the machine
 * being run on.
 * 
 * Compile with:
 *   nvcc -o dq00 dq00.cu
 * 
 * Dr Kevan Buckley, University of Wolverhampton, 2018 
 ****************************************************************************/
int main() {
  int device_count;
  hipError_t error_id = hipGetDeviceCount(&device_count);

  if (error_id != 0) {
    fprintf(stderr, "hipGetDeviceCount returned %d\n-> %s\n", 
           (int)error_id, hipGetErrorString(error_id));
    exit(1);
  }

  if (device_count == 0) {
    fprintf(stderr, "There are no available device(s) that support CUDA\n");
  } else {
    printf("Detected %d CUDA capable device(s)\n", device_count);
  }  
  
  return 0;
}
